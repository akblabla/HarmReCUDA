#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "LinearAlgebraStructs.h"

__global__
void findHighestEnergyIndices_kernel(matrix dest_d, matrix src_d, unsigned int rowFactor, unsigned int columnFactor)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int destColumnIndex = (i / dest_d.rows) % dest_d.columns;
	int destRowIndex = i % dest_d.rows;
	int srcColumnIndex = ((i / columnFactor) / src_d.rows) % src_d.columns;
	int srcRowIndex = ((i / rowFactor) % src_d.rows);
	if (destColumnIndex < dest_d.columns && srcColumnIndex < src_d.columns) { //don't write outside matrix
		dest_d.elements[MATRIX_INDEX(destRowIndex, destColumnIndex, dest_d.rows)] *= src_d.elements[MATRIX_INDEX(srcRowIndex, srcColumnIndex, src_d.rows)];
	}
}


extern "C" void findHighestEnergiesFrequencies_cuda(matrix energies_d, matrix centralFreq_d, const matrix src_d) {
	int N = energies_d.rows * energies_d.columns;
	findHighestEnergyIndices_kernel << <(N + 1023) / 1024, 1024 >> > (energies_d, src_d, 1, 1);
}