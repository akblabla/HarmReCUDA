#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "GenerateProjectionMatrix_d.h"

__global__
void generateProjectionMatrix_kernel(matrix dest_d, const double minFreq, const double maxFreq, const double startTime, const double deltaTime, const matrix harmonics_d)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int columnIndex = i/dest_d.rows;
	int rowIndex = ((2 * i) % dest_d.rows); //skip every second row
	double t = columnIndex*deltaTime+startTime;
	int fundamentalFrequencyIndex = rowIndex / (harmonics_d.rows);
	int harmonicIndex = (rowIndex) % (harmonics_d.rows);
	double fundamentalFrequency = (minFreq + (fundamentalFrequencyIndex) * (maxFreq - minFreq) / dest_d.columns);
	double freq = fundamentalFrequency * harmonics_d.elements[harmonicIndex];
	
	
	double phase = t*freq;
	if (columnIndex < dest_d.columns) //make sure not to write outside of matrix, incase the number of elements did not have a base of 1024
	sincos(
		phase,
		&(dest_d.elements[dest_d.rows * columnIndex + rowIndex+1]),
		&(dest_d.elements[dest_d.rows * columnIndex + rowIndex])
	);
}
/**
*creates a projection matrix on the gpu to the given matrix on the device.
**/
extern "C" void generateProjectionMatrix_cuda(matrix dest_d, const double minFreq, const double maxFreq, const double startTime, const double deltaTime, const matrix harmonics_d){
	int N = dest_d.rows* dest_d.columns/2; //each thread handles two elements of the matrix
	generateProjectionMatrix_kernel<<<(N+ 1023)/ 1024, 1024 >>>(dest_d, minFreq, maxFreq, startTime, deltaTime, harmonics_d);
}