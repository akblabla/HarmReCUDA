#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#define M_PI   3.14159265358979323846264338327950288
#include "hipblas.h"
#include "LinearAlgebraStructs.h"

__global__
void generateProjectionMatrix_kernel(matrix dest_d, const matrix freq_d, const matrix time_d, const matrix harmonics_d)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int columnIndex = (2 * i)/dest_d.rows;
	int rowIndex = ((2 * i) % dest_d.rows); //skip every second row
	double t = time_d.elements[columnIndex];
	int fundamentalFrequencyIndex = rowIndex / (2*harmonics_d.rows);
	int harmonicIndex = (rowIndex/2) % (harmonics_d.rows);
	double fundamentalFrequency = 2 * M_PI *freq_d.elements[fundamentalFrequencyIndex];
	double freq = fundamentalFrequency * harmonics_d.elements[harmonicIndex];
	double phase = freq* t;
	if (columnIndex < dest_d.columns) {//make sure not to write outside of matrix, incase the number of elements did not have a base of 1024
		sincos(
			phase,
			&(dest_d.elements[MATRIX_INDEX(rowIndex + 1, columnIndex, dest_d.ld)]),
			&(dest_d.elements[MATRIX_INDEX(rowIndex, columnIndex, dest_d.ld)])
		);
		//dest_d.elements[MATRIX_INDEX(rowIndex + 1, columnIndex, dest_d.rows)] *= renomalizeFactor;
		//dest_d.elements[MATRIX_INDEX(rowIndex, columnIndex, dest_d.rows)] *= renomalizeFactor;
	}
}
/**
*creates a projection matrix on the gpu to the given matrix on the device.
**/
extern "C" void generateProjectionMatrix_cuda(matrix dest_d, const matrix freq_d, const matrix time_d, const matrix harmonics_d){
	int N = dest_d.rows* dest_d.columns/2; //each thread handles two elements of the matrix
	generateProjectionMatrix_kernel <<< (N+ 1023)/ 1024, 1024 >>>(dest_d, freq_d, time_d, harmonics_d);
}