#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "LinearAlgebraStructs.h"

__global__
void elementWiseMultiplication_kernel(matrix dest_d, matrix src_d, unsigned int rowFactor, unsigned int columnFactor)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int destColumnIndex = i / dest_d.rows;
	int destRowIndex = i % dest_d.rows;
	if (destColumnIndex < dest_d.columns) { //don't write outside matrix
		dest_d.elements[MATRIX_INDEX(destRowIndex, destColumnIndex, dest_d.rows)] *= src_d.elements[MATRIX_INDEX(destRowIndex / rowFactor, destColumnIndex / columnFactor, src_d.rows)];
	}
}
/**
*Multiplies element wise the src matrix and the dest matrix and stores the result in dest. rowFactor and columnFactor expands the src matrix by the given factor.
**/
extern "C" void elementWiseMultiplication_cuda(matrix dest_d, const matrix src_d, unsigned int rowFactor, unsigned int columnFactor) {
	int N = dest_d.rows * dest_d.columns;
	elementWiseMultiplication_kernel << <(N + 1023) / 1024, 1024 >> > (dest_d, src_d, rowFactor, columnFactor);
}