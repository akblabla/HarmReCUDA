#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "LinearAlgebraStructs.h"

__global__
void elementWiseMultiplication_kernel(matrix dest_d, matrix src_d, unsigned int rowFactor, unsigned int columnFactor)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int destColumnIndex = (i / dest_d.rows)%dest_d.columns;
	int destRowIndex = i % dest_d.rows;
	int srcColumnIndex = ((i / rowFactor / columnFactor) / src_d.rows)% src_d.columns;
	int srcRowIndex = ((i / rowFactor) % src_d.rows);
	if (destColumnIndex < dest_d.columns && srcColumnIndex < src_d.columns) { //don't write outside matrix
		dest_d.elements[MATRIX_INDEX(destRowIndex, destColumnIndex, dest_d.rows)] *= src_d.elements[MATRIX_INDEX(srcRowIndex, srcColumnIndex, src_d.rows)];
	}
}
/**
*Multiplies element wise the src matrix and the dest matrix and stores the result in dest. rowFactor and columnFactor expands the src matrix by the given factor. If src matrix is smaller than dest after being expanded, the index wraps.
**/
extern "C" void elementWiseMultiplication_cuda(matrix dest_d, const matrix src_d, unsigned int rowFactor, unsigned int columnFactor) {
	int N = dest_d.rows * dest_d.columns;
	elementWiseMultiplication_kernel << <(N + 1023) / 1024, 1024 >> > (dest_d, src_d, rowFactor, columnFactor);
}